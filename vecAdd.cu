#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

using namespace std;

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double* a, double* b, double* c, int width)
{
	// Get our global thread ID
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	// Make sure we do not go out of bounds
	if (id < width * width) {
		c[id] = a[id] + b[id];
	}

}

__global__ void vecAddOneRow(double* a, double* b, double* c, int width) {

	int id = threadIdx.x;
	int k = 0;
	double value;
	// Make sure we do not go out of bounds
	for (k = 0; k < width ; k++) {
		value = a[id * width + k] + b[id * width + k];
		c[id * width + k] = value;
	}

}

__global__ void vecAddOneCol(double* a, double* b, double* c, int width) {

	int id = threadIdx.x;
	int k = 0;
	double value;
	// Make sure we do not go out of bounds
	for (k = 0; k < width; k++) {
		value = a[k* width + id] + b[k * width + id];
		c[k * width + id] = value;
	}

}


int main(int argc, char* argv[])
{
	// Size of vectors
	int n = 1024 * 1024;
	int width = 1024;

	// Host input vectors
	double* h_a;
	double* h_b;
	//Host output vector
	double* h_c;

	// Device input vectors
	double* d_a;
	double* d_b;
	//Device output vector
	double* d_c;

	// Size, in bytes, of each vector
	size_t bytes = n * sizeof(double);

	// Allocate memory for each vector on host
	h_a = (double*)malloc(bytes);
	h_b = (double*)malloc(bytes);
	h_c = (double*)malloc(bytes);

	// Allocate memory for each vector on GPU
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	int i;
	// Initialize vectors on host
	for (i = 0; i < n; i++) {
		h_a[i] = sin(i) * sin(i);
		h_b[i] = cos(i) * cos(i);
	}

	// Copy host vectors to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int blockSize, gridSize;

	// Number of threads in each thread block
	blockSize = 1024;

	// Number of thread blocks in grid
	gridSize = (int)ceil((float)n / blockSize);
	
	vecAdd << <gridSize, blockSize >> > (d_a, d_b, d_c, width);

	// Copy array back to host
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Sum up vector c and print result divided by n, this should equal 1 within error
	double sum = 0;

	for (int j = 0; j < 5; j++) {
		for (i = 0; i < 5; i++) {
			printf("%f  ", h_c[j*width+i]);
		}
		printf("\n");
	}


		// Release device memory
	    hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

		// Release host memory
		free(h_a);
		free(h_b);
		free(h_c);

		return 0;
	}
